#include "hip/hip_runtime.h"
#include "Cudamatrix3x3.h"

#include <iostream>
#include <cmath>

using namespace std;

namespace CGL {

  double& CudaMatrix3x3::operator()( int i, int j ) {
    return entries[j][i];
  }

  const double& CudaMatrix3x3::operator()( int i, int j ) const {
    return entries[j][i];
  }

  Vector3D& CudaMatrix3x3::operator[]( int j ) {
      return entries[j];
  }

  const Vector3D& CudaMatrix3x3::operator[]( int j ) const {
    return entries[j];
  }

  void CudaMatrix3x3::zero( double val ) {
    // sets all elements to val
    entries[0] = entries[1] = entries[2] = Vector3D( val, val, val );
  }

  double CudaMatrix3x3::det( void ) const {
    const CudaMatrix3x3& A( *this );

    return -A(0,2)*A(1,1)*A(2,0) + A(0,1)*A(1,2)*A(2,0) +
            A(0,2)*A(1,0)*A(2,1) - A(0,0)*A(1,2)*A(2,1) -
            A(0,1)*A(1,0)*A(2,2) + A(0,0)*A(1,1)*A(2,2) ;
  }

  double CudaMatrix3x3::norm( void ) const {
    return sqrt( entries[0].norm2() +
                 entries[1].norm2() +
                 entries[2].norm2() );
  }

  CudaMatrix3x3 CudaMatrix3x3::operator-( void ) const {

   // returns -A
    const CudaMatrix3x3& A( *this );
    CudaMatrix3x3 B;

    B(0,0) = -A(0,0); B(0,1) = -A(0,1); B(0,2) = -A(0,2);
    B(1,0) = -A(1,0); B(1,1) = -A(1,1); B(1,2) = -A(1,2);
    B(2,0) = -A(2,0); B(2,1) = -A(2,1); B(2,2) = -A(2,2);

    return B;
  }

  void CudaMatrix3x3::operator+=( const CudaMatrix3x3& B ) {

    CudaMatrix3x3& A( *this );
    double* Aij = (double*) &A;
    const double* Bij = (const double*) &B;

    *Aij++ += *Bij++;
    *Aij++ += *Bij++;
    *Aij++ += *Bij++;
    *Aij++ += *Bij++;
    *Aij++ += *Bij++;
    *Aij++ += *Bij++;
    *Aij++ += *Bij++;
    *Aij++ += *Bij++;
    *Aij++ += *Bij++;
  }

  CudaMatrix3x3 CudaMatrix3x3::operator-( const CudaMatrix3x3& B ) const {
    const CudaMatrix3x3& A( *this );
    CudaMatrix3x3 C;

    for( int i = 0; i < 3; i++ )
    for( int j = 0; j < 3; j++ )
    {
       C(i,j) = A(i,j) - B(i,j);
    }

    return C;
  }

  CudaMatrix3x3 CudaMatrix3x3::operator*( double c ) const {
    const CudaMatrix3x3& A( *this );
    CudaMatrix3x3 B;

    for( int i = 0; i < 3; i++ )
    for( int j = 0; j < 3; j++ )
    {
       B(i,j) = c*A(i,j);
    }

    return B;
  }

  CudaMatrix3x3 operator*( double c, const CudaMatrix3x3& A ) {

    CudaMatrix3x3 cA;
    const double* Aij = (const double*) &A;
    double* cAij = (double*) &cA;

    *cAij++ = c * (*Aij++);
    *cAij++ = c * (*Aij++);
    *cAij++ = c * (*Aij++);
    *cAij++ = c * (*Aij++);
    *cAij++ = c * (*Aij++);
    *cAij++ = c * (*Aij++);
    *cAij++ = c * (*Aij++);
    *cAij++ = c * (*Aij++);
    *cAij++ = c * (*Aij++);

    return cA;
  }

  CudaMatrix3x3 CudaMatrix3x3::operator*( const CudaMatrix3x3& B ) const {
    const CudaMatrix3x3& A( *this );
    CudaMatrix3x3 C;

    for( int i = 0; i < 3; i++ )
    for( int j = 0; j < 3; j++ )
    {
       C(i,j) = 0.;

       for( int k = 0; k < 3; k++ )
       {
          C(i,j) += A(i,k)*B(k,j);
       }
    }

    return C;
  }

  Vector3D CudaMatrix3x3::operator*( const Vector3D& x ) const {
    return x[0]*entries[0] +
           x[1]*entries[1] +
           x[2]*entries[2] ;
  }

  CudaMatrix3x3 CudaMatrix3x3::T( void ) const {
    const CudaMatrix3x3& A( *this );
    CudaMatrix3x3 B;

    for( int i = 0; i < 3; i++ )
    for( int j = 0; j < 3; j++ )
    {
       B(i,j) = A(j,i);
    }

    return B;
  }

  CudaMatrix3x3 CudaMatrix3x3::inv( void ) const {
    const CudaMatrix3x3& A( *this );
    CudaMatrix3x3 B;

    B(0,0) = -A(1,2)*A(2,1) + A(1,1)*A(2,2); B(0,1) =  A(0,2)*A(2,1) - A(0,1)*A(2,2); B(0,2) = -A(0,2)*A(1,1) + A(0,1)*A(1,2);
    B(1,0) =  A(1,2)*A(2,0) - A(1,0)*A(2,2); B(1,1) = -A(0,2)*A(2,0) + A(0,0)*A(2,2); B(1,2) =  A(0,2)*A(1,0) - A(0,0)*A(1,2);
    B(2,0) = -A(1,1)*A(2,0) + A(1,0)*A(2,1); B(2,1) =  A(0,1)*A(2,0) - A(0,0)*A(2,1); B(2,2) = -A(0,1)*A(1,0) + A(0,0)*A(1,1);

    B /= det();

    return B;
  }

  void CudaMatrix3x3::operator/=( double x ) {
    CudaMatrix3x3& A( *this );
    double rx = 1./x;

    for( int i = 0; i < 3; i++ )
    for( int j = 0; j < 3; j++ )
    {
       A( i, j ) *= rx;
    }
  }

  CudaMatrix3x3 CudaMatrix3x3::identity( void ) {
    CudaMatrix3x3 B;

    B(0,0) = 1.; B(0,1) = 0.; B(0,2) = 0.;
    B(1,0) = 0.; B(1,1) = 1.; B(1,2) = 0.;
    B(2,0) = 0.; B(2,1) = 0.; B(2,2) = 1.;

    return B;
  }

  CudaMatrix3x3 CudaMatrix3x3::crossProduct( const Vector3D& u ) {
    CudaMatrix3x3 B;

    B(0,0) =   0.;  B(0,1) = -u.z;  B(0,2) =  u.y;
    B(1,0) =  u.z;  B(1,1) =   0.;  B(1,2) = -u.x;
    B(2,0) = -u.y;  B(2,1) =  u.x;  B(2,2) =   0.;

    return B;
  }

  CudaMatrix3x3 outer( const Vector3D& u, const Vector3D& v ) {
    CudaMatrix3x3 B;
    double* Bij = (double*) &B;

    *Bij++ = u.x*v.x;
    *Bij++ = u.y*v.x;
    *Bij++ = u.z*v.x;
    *Bij++ = u.x*v.y;
    *Bij++ = u.y*v.y;
    *Bij++ = u.z*v.y;
    *Bij++ = u.x*v.z;
    *Bij++ = u.y*v.z;
    *Bij++ = u.z*v.z;

    return B;
  }

  std::ostream& operator<<( std::ostream& os, const CudaMatrix3x3& A ) {
    for( int i = 0; i < 3; i++ )
    {
       os << "[ ";

       for( int j = 0; j < 3; j++ )
       {
          os << A(i,j) << " ";
       }

       os << "]" << std::endl;
    }

    return os;
  }

  Vector3D& CudaMatrix3x3::column( int i ) {
    return entries[i];
  }

  const Vector3D& CudaMatrix3x3::column( int i ) const {
    return entries[i];
  }
}
