#include "hip/hip_runtime.h"
#include "cudapathtracer.h"

namespace CGL {

/*   
*   
*
*/
__global__ void raytrace_cuda_pixel_helper(size_t* x, size_t* y, Spectrum* sp){
	sp->r = 0.30;
	sp->g = 0.40;
	sp->b = 0.050;
}

Spectrum raytrace_cuda_pixel(size_t x, size_t y){
	//PUT CUDA MALLOC STUFF HERE
	//size_t *host_x, *host_y; //Host xy
	size_t *dev_x, *dev_y; //Device x y
	Spectrum *dev_sp;
	int size_tsize = sizeof(size_t);
	//malloc host_x, host_y
	//host_x = malloc(size_tsize);
	// = malloc(size_tsize);

	//cudamalloc x, y, spectrum
	hipError_t err = hipMalloc((void **) &dev_x, size_tsize);
	if (err != hipSuccess){
		printf("%s1\n", hipGetErrorString(err));
	}
	err = hipMalloc((void **) &dev_y, size_tsize);
		if (err != hipSuccess){
		printf("%s2\n", hipGetErrorString(err));
	}
	err = hipMalloc((void **) &dev_sp, sizeof(Spectrum));
	if (err != hipSuccess){
		printf("%s3\n", hipGetErrorString(err));
	}
	//cudaMemCpy
	err = hipMemcpy(dev_x, &x, sizeof(size_t), hipMemcpyHostToDevice);
	if (err != hipSuccess){
		printf("%s4\n", hipGetErrorString(err));
	}
	err = hipMemcpy(dev_y, &y, sizeof(size_t), hipMemcpyHostToDevice);
	if (err != hipSuccess){
		printf("%s5\n", hipGetErrorString(err));
	}
	//Call helper
	raytrace_cuda_pixel_helper<<<1,1>>>(dev_x, dev_y, dev_sp);

	//Copy Result
	Spectrum *result = (Spectrum *)malloc(sizeof(Spectrum));

	err = hipMemcpy(result, dev_sp, sizeof(Spectrum), hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		printf("%s\n", hipGetErrorString(err));
	}
	//Cleanup - DON'T FORGET TO UN-MALLOC FREEE IT
	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_sp);
	//free(host_x);
	//free(host_y);
	return *result;
}

/*
Spectrum PathTracer::raytrace_pixel(size_t x, size_t y) {

  // Part 1, Task 1:
  // Make a loop that generates num_samples camera rays and traces them 
  // through the scene. Return the average Spectrum. 

  int num_samples = ns_aa; // total samples to evaluate
  Vector2D origin = Vector2D(x,y); // bottom left corner of the pixel
  Spectrum average = Spectrum();
  //Loop, for number of samples, get the color
  Vector2D sampler = Vector2D(0.5, 0.5); //First pixel is always 0.5
  for(int i = 0; i < num_samples; i++){
    Vector2D point = Vector2D(((double)x + sampler.x)/sampleBuffer.w, ((double)y + sampler.y)/sampleBuffer.h);
    Ray r = camera->generate_ray(point.x, point.y);
    r.depth = max_ray_depth;
    average += trace_ray(r, true);

    sampler = gridSampler->get_sample(); //For next iteration
    
  }

}

*/

} //namespace CGL