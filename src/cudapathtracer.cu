#include "hip/hip_runtime.h"
#include "cudapathtracer.h"
namespace CGL {
/*   
*   
*
*/
__global__ void raytrace_cuda_pixel_helper(size_t* x, size_t* y, Spectrum* sp, struct data_necessary* cuda_data){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    sp[i].r = 0.20;
    sp[i].g = 0.40;
    sp[i].b = 0.080;

  	int num_samples = *cuda_data->ns_aa; // total samples to evaluate
	HDRImageBuffer* sampleBuffer = cuda_data->sampleBuffer;
  	Camera* camera = cuda_data->camera;
  	
  	CudaVector2D origin = CudaVector2D(*x,*y); // bottom left corner of the pixel
  	CudaSpectrum average = CudaSpectrum();
  //Loop, for number of samples, get the color
  	CudaVector2D sampler = CudaVector2D(0.5, 0.5); //First pixel is always 0.5

  	for(int i = 0; i < num_samples; i++){
	    CudaVector2D point = CudaVector2D(((double)*x + sampler.x)/sampleBuffer->w, ((double)*y + sampler.y)/sampleBuffer->h);
	    CudaRay r = path_cuda_generate_ray(point.x, point.y);
	    r.depth = *cuda_data->max_ray_depth;
	    //average += trace_cuda_ray(r, true, cuda_data);

	    //sampler = cuda_data->gridSampler->get_sample(); //For next iteration
	    
  }
  // Part 1, Task 1:
  // Make a loop that generates num_samples camera rays and traces them 
  // through the scene. Return the average Spectrum. 
/*
  for(int i = 0; i < num_samples; i++){
    CudaVector2D point = CudaVector2D(((double)*x + sampler.x)/sampleBuffer->w, ((double)*y + sampler.y)/sampleBuffer->h);
    CudaRay r = camera->cuda_generate_ray(point.x, point.y);
    r.depth = *cuda_data->max_ray_depth;
    average += trace_cuda_ray(r, true, cuda_data);

    sampler = cuda_data->gridSampler->get_sample(); //For next iteration
    
  }
*/

}

//Originally from ccamera.cpp
__device__ CudaRay path_cuda_generate_ray(double x, double y){
  // Part 1, Task 2:
  // compute position of the input sensor sample coordinate on the
  // canonical sensor plane one unit away from the pinhole.
  // Note: hFov and vFov are in degrees.
  // 
  /*
  double hFovRad = *cudahFov * (PI / 180);
  double vFovRad = *cudavFov * (PI / 180);
  CudaVector3D lower_left  = CudaVector3D(-tan(hFovRad*.5), -tan(vFovRad*.5),-1);
  CudaVector3D upper_right = CudaVector3D( tan(hFovRad*.5),  tan(vFovRad*.5),-1);
  CudaVector3D direction = CudaVector3D(lower_left.x + x*(upper_right.x - lower_left.x), 
                                 lower_left.y + y*(upper_right.y - lower_left.y),
                                    -1 );
  direction = *cudac2w*direction;
  direction.normalize();
  
  CudaRay my_ray = CudaRay(*hipPos, direction);
  my_ray.min_t = *cudaNClip;
  my_ray.max_t = *cudaFClip;
  */
  return CudaRay(CudaVector3D(0, 0, 0), CudaVector3D(0, 0, 0)); //my_ray;


}


void hipMemcpy(size_t* ns_aa, HDRImageBuffer* sampleBuffer, Camera* c){
	// hipMalloc((void **) &cuda_ns_aa, sizeof(size_t));
 //    hipMemcpy(cuda_ns_aa, ns_aa, sizeof(size_t), hipMemcpyHostToDevice);

 //    hipMalloc((void **) &cuda_sampleBuffer, sizeof(HDRImageBuffer));
 //    hipMemcpy(cuda_sampleBuffer, sampleBuffer, sizeof(HDRImageBuffer), hipMemcpyHostToDevice);

 //    hipMalloc((void **) &cuda_c, sizeof(Camera));
 //    hipMemcpy(cuda_c, c, sizeof(Camera), hipMemcpyHostToDevice);

}

__global__ void instantiate_Necesary(struct data_necessary* data){
	//BVH Instantiation Here
}

//Returns struct with all CUDA pointers
struct data_necessary* cudaMallocNecessary(struct host_data_necessary* data){
	printf("Starting hipMalloc\n");
    struct data_necessary* host_data = (struct data_necessary*) malloc(sizeof(struct data_necessary));
    struct data_necessary* cuda_data;
    hipMalloc((void **) &cuda_data, sizeof(struct data_necessary));

    size_t* ns_aa;
    hipMalloc((void **) &ns_aa, sizeof(size_t));
    hipMemcpy(ns_aa, data->ns_aa, sizeof(size_t), hipMemcpyHostToDevice);
    host_data->ns_aa = ns_aa;

    HDRImageBuffer *sampleBuffer;
    hipMalloc((void **) &sampleBuffer, sizeof(HDRImageBuffer));
    hipMemcpy(sampleBuffer, data->sampleBuffer, sizeof(HDRImageBuffer), hipMemcpyHostToDevice);
    host_data->sampleBuffer = sampleBuffer;

    Camera *camera;
    hipMalloc((void **) &camera, sizeof(Camera));
    hipMemcpy(camera, data->camera, sizeof(Camera), hipMemcpyHostToDevice);
    host_data->camera = camera;

    size_t* max_ray_depth;
    hipMalloc((void **) &max_ray_depth, sizeof(size_t));
    hipMemcpy(max_ray_depth, data->max_ray_depth, sizeof(size_t), hipMemcpyHostToDevice);
    host_data->max_ray_depth = max_ray_depth;

    CudaSampler2D *gridSampler;
    hipMalloc((void **) &gridSampler, sizeof(CudaSampler2D));
    hipMemcpy(gridSampler, data->gridSampler, sizeof(CudaSampler2D), hipMemcpyHostToDevice);
    host_data->gridSampler = gridSampler;
    printf("Ending hipMalloc\n");
    
    hipMemcpy(cuda_data, host_data, sizeof(struct data_necessary), hipMemcpyHostToDevice);
    return cuda_data;
}

void cudaFreeNecessary(struct data_necessary* cuda_data){
printf("Starting hipMalloc\n");
    
    hipFree(cuda_data->ns_aa);

    hipFree(cuda_data->sampleBuffer);

    hipFree(cuda_data->camera);

    hipFree(cuda_data->max_ray_depth);

    hipFree(cuda_data->gridSampler);
    printf("Ending hipMalloc\n");
 
    hipFree(cuda_data);
}
void raytrace_cuda_tile(int tile_x, int tile_y,
                                int tile_w, int tile_h, HDRImageBuffer *sampleBuffer,
                                size_t imageTileSize, vector<int> *tile_samples,
                                ImageBuffer *frameBuffer, struct host_data_necessary *data) {

    struct data_necessary* cuda_data = cudaMallocNecessary(data);
    
    hipError_t err = hipSetDevice(0);
    if(err != hipSuccess){
    	printf("err not success\n");
    }
	size_t w = sampleBuffer->w;
    size_t h = sampleBuffer->h;

    size_t num_tiles_w = w / imageTileSize + 1;

    size_t tile_start_x = tile_x;
    size_t tile_start_y = tile_y;

    size_t tile_end_x = std::min(tile_start_x + tile_w, w);
    size_t tile_end_y = std::min(tile_start_y + tile_h, h);

    size_t tile_idx_x = tile_x / imageTileSize;
    size_t tile_idx_y = tile_y / imageTileSize;
    size_t num_samples_tile = (*tile_samples)[tile_idx_x + tile_idx_y * num_tiles_w];

    size_t *host_x, *host_y;
    size_t *dev_x, *dev_y;
    Spectrum *dev_sp;

    size_t tile_length_x = tile_end_x - tile_start_x;
    size_t tile_length_y = tile_end_y - tile_start_y;

    host_x = (size_t *)malloc(sizeof(size_t) * tile_length_x);
    host_y = (size_t *)malloc(sizeof(size_t) * tile_length_y);

    for (size_t y = 0; y < tile_length_y; y++) {
        host_y[y] = tile_start_y + y;
    }
    for (size_t x = 0; x < tile_length_x; x++) {
        host_x[x] = tile_start_x + x;
    }

    //cudamalloc x, y, spectrum
    hipMalloc((void **) &dev_x, sizeof(size_t) * tile_length_x);
    hipMalloc((void **) &dev_y, sizeof(size_t) * tile_length_y);
    hipMalloc((void **) &dev_sp, sizeof(Spectrum) * tile_length_x * tile_length_y);

    //cudaMemCpy
    hipMemcpy(dev_x, &host_x, sizeof(size_t) * tile_length_x, hipMemcpyHostToDevice);
    hipMemcpy(dev_y, &host_y, sizeof(size_t) * tile_length_y, hipMemcpyHostToDevice);
    
    int N = tile_length_x;
    int M = tile_length_y;

    //Call helper
    raytrace_cuda_pixel_helper<<<N,M>>>(dev_x, dev_y, dev_sp, cuda_data);
    hipDeviceSynchronize();
    //Copy Result
    Spectrum *result = (Spectrum *)malloc(sizeof(Spectrum) * tile_length_x * tile_length_y);

    hipMemcpy(result, dev_sp, (sizeof(Spectrum) * tile_length_x * tile_length_y), hipMemcpyDeviceToHost);
    
    for (size_t x = 0; x < tile_length_x; x++) {
        //if (!continueRaytracing) return;
        for (size_t y = 0; y < tile_length_y; y++) {
            sampleBuffer->update_pixel(result[x * tile_length_x + y], tile_start_x + x, tile_start_y + y);
        }
    }
   

    //Cleanup - DON'T FORGET TO UN-MALLOC FREE IT
    free(host_x);
    free(host_y);
    free(result);
    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_sp);
   // cudaFreeNecessary(cuda_data);

    (*tile_samples)[tile_idx_x + tile_idx_y * num_tiles_w] += 1;
    sampleBuffer->toColor(*frameBuffer, tile_start_x, tile_start_y, tile_end_x, tile_end_y);
}



void testblahlah() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  Max Threads Per Block: %d\n",
           prop.maxThreadsPerBlock);
    printf("  Multiprocessor Count: %d\n\n",
           prop.multiProcessorCount);
  }
}

__device__ CudaSpectrum trace_cuda_ray( CudaRay &r, bool includeLe, struct data_necessary* cuda_data) {


  CudaIntersection isect;
  CudaSpectrum L_out;
  printf("hai\n");
  // You will extend this in part 2. 
  // If no intersection occurs, we simply return black.
  // This changes if you implement hemispherical lighting for extra credit.
  if (!cuda_data->bvh->intersect(r, &isect)) 
    return L_out;

  // This line returns a color depending only on the normal vector 
  // to the surface at the intersection point.
  // Remove it when you are ready to begin Part 3.
  //return normal_shading(isect.n);

  // We only include the emitted light if the previous BSDF was a delta distribution
  // or if the previous ray came from the camera.
  if (includeLe)
    L_out += isect.bsdf->get_emission();

  // You will implement this in part 3. 
  // Delta BSDFs have no direct lighting since they are zero with probability 1 --
  // their values get accumulated through indirect lighting, where the BSDF 
  // gets to sample itself.
  if (!isect.bsdf->is_delta()) 
 //   L_out += estimate_direct_lighting(r, isect, cuda_data);
  // You will implement this in part 4.
  // If the ray's depth is zero, then the path must terminate
  // and no further indirect lighting is calculated.
  if (r.depth > 0)
 //   L_out += estimate_indirect_lighting(r, isect, cuda_data);
  
  return L_out;

}
__device__ CudaSpectrum estimate_direct_lighting( CudaRay& r,  CudaIntersection& isect, struct data_necessary* cuda_data) {

// TODO Part 3
/*
  // make a coordinate system for a hit point
  // with N aligned with the Z direction.
  CudaMatrix3x3 o2w;
  make_cuda_coord_space(o2w, isect.n);
  CudaMatrix3x3 w2o = o2w.T();

  // w_out points towards the source of the ray (e.g.,
  // toward the camera if this is a primary ray)
  const CudaVector3D& hit_p = r.o + r.d * isect.t;
  const CudaVector3D& w_out = w2o * (-r.d);

  CudaSpectrum L_out = CudaSpectrum();
  for (int j = 0; j < cuda_data->scene->lights_len; j++){
    CudaSceneLight* light = cuda_data->scene->lights[j];
    int num_samples = 1;
    if(light->is_delta_light() == false){//Check if delta light.
      //If yes, ask for one sample
      num_samples = *cuda_data->ns_area_light;
    }
    CudaSpectrum sample_out = CudaSpectrum();
    for(int i = 0; i < num_samples; i++){
        CudaVector3D wi = CudaVector3D();
        float distToLight = 0;
        float pdf = 0;
        CudaSpectrum rad_in = light->sample_L(hit_p, &wi,
                                            &distToLight,
                                            &pdf); //incoming radiance
        CudaVector3D w_in = w2o*wi; //Object space vector
        if(w_in.z > 0){  
          CudaVector3D direction = wi;
          direction.normalize();
          CudaRay shadow = CudaRay(hit_p + EPS_D*direction, direction);
          shadow.max_t = distToLight;
          if(!cuda_data->bvh->intersect(shadow)){
            CudaSpectrum s = isect.bsdf->f(w_out, w_in); //local space
            sample_out += s * (rad_in * fabs(w_in.z))/ pdf;
          }

        }

    }
    L_out += sample_out/num_samples;
  }
  */
  return CudaSpectrum(0, 0, 0); //L_out;



}


} //namespace
